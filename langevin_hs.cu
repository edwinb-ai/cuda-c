#include "hip/hip_runtime.h"
#include "functionsl.h"

int main(int argc, char const *argv[])
{
    // Definir la GPU
    hipSetDevice(0);

    // Archivos para trabajar
    FILE *f_iniconf;
    FILE *f_gr;
    FILE *f_final;
    FILE *f_ener;
    FILE *wt_f;

    //  Numero de particulas
    int simple_part = 12;
    int n_part = simple_part*simple_part*simple_part;
    //  Fracción de empaquetamiento
    float phi = atof(argv[1]);
    //  Densidad
    float rho = 6.0 * phi / PI;
    //  Configuraciones para termalizar
    int nct = atoi(argv[2]);
    //  Termalización
    int ncp = atoi(argv[3]);
    //  Paso de tiempo
    float d_tiempo = atof(argv[4]);
    unsigned long long int seed = (unsigned long long int)atoi(argv[5]);
    //  Revisar si ya se tiene una configuración de termalización
    int config_termal = atoi(argv[6]);

    // Tamaño de caja
    float l_caja = powf((float)(n_part) / rho, 1.0 / 3.0);
    float radio_c = l_caja / 2.0;
    float dr = radio_c / nm;

    // Mostrar información del sistema
    printf("El tamaño de la caja es: %f\n", l_caja);
    printf("Distancia media entre partículas: %f\n", powf(rho, -1.0 / 3.0));
    printf("Radio de corte: %f\n", radio_c);

    // ! RNG variables
    hiprandGenerator_t gen;
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
    hiprandSetPseudoRandomGeneratorSeed(gen, seed);
    float *rngvec_dev;
    int rng_size = (int)(3 * n_part);
    hipMallocManaged(&rngvec_dev, rng_size * sizeof(float));

    // Inicializar los arreglos
    float *x;
    hipMallocManaged(&x, n_part * sizeof(float));
    float *y;
    hipMallocManaged(&y, n_part * sizeof(float));
    float *z;
    hipMallocManaged(&z, n_part * sizeof(float));
    float *fx;
    hipMallocManaged(&fx, n_part * sizeof(float));
    float *fy;
    hipMallocManaged(&fy, n_part * sizeof(float));
    float *fz;
    hipMallocManaged(&fz, n_part * sizeof(float));
    float *g;
    hipMallocManaged(&g, nm * sizeof(float));
    float *t;
    hipMallocManaged(&t, mt_n * sizeof(float));
    float *wt;
    hipMallocManaged(&wt, mt_n * sizeof(float));

    float *cfx;
    hipMallocManaged(&cfx, mt_n * n_part * sizeof(float));
    float *cfy;
    hipMallocManaged(&cfy, mt_n * n_part * sizeof(float));
    float *cfz;
    hipMallocManaged(&cfz, mt_n * n_part * sizeof(float));
    float *ener;
    hipMallocManaged(&ener, n_part * sizeof(float));
    float total_ener = 0.0f;

    // Asignar hilos y bloques
    int hilos = 256;
    int bloques = roundf((n_part + hilos - 1) / hilos);

    // SI SE INGRESA UNA CONFIGURACION DE TERMALIZACION, SE LEE:
    if (config_termal == 1)
    {
        printf("Se va a leer datos existentes de termalización.\n");
        f_final = fopen("final_conf.dat", "r");
        for (int i = 0; i < n_part; i++)
        {
            fscanf(f_final, "%f", &x[i]);
            fscanf(f_final, "%f", &y[i]);
            fscanf(f_final, "%f", &z[i]);
            fscanf(f_final, "%f", &fx[i]);
            fscanf(f_final, "%f", &fy[i]);
            fscanf(f_final, "%f", &fz[i]);
        }
        fclose(f_final);
    }
    // SI NO SE CUENTA CON UNA, HAY QUE CREARLA:
    else
    {
        // Configuración inicial
        iniconf(x, y, z, rho, radio_c, n_part);
        f_iniconf = fopen("conf_inicial.dat", "w");
        for (int i = 0; i < n_part; i++)
        {
            fprintf(f_iniconf, "%.10f %.10f %.10f\n", x[i], y[i], z[i]);
        }
        fclose(f_iniconf);

        // Verificar que la energía es cero
        rdf_force<<<bloques, hilos>>>(x, y, z, fx, fy, fz, n_part, l_caja, ener);
        hipDeviceSynchronize();
        total_ener = 0.0f;
        for (int i = 0; i < n_part; i++)
            total_ener += ener[i];
        printf("E/N: %.10f\n", total_ener / ((float)(n_part)));

        // Termalizar el sistema
        f_ener = fopen("energia.dat", "w");
        f_final = fopen("final_conf.dat", "w");

        for (int i = 0; i < nct; i++)
        {
            // * Crear números aleatorios
            hiprandGenerateNormal(gen, rngvec_dev, rng_size, 0.0f, 1.0f);
            position<<<bloques, hilos>>>(x, y, z, fx, fy, fz, d_tiempo, l_caja, n_part, 1, rngvec_dev);
            hipDeviceSynchronize();
            rdf_force<<<bloques, hilos>>>(x, y, z, fx, fy, fz, n_part, l_caja, ener);
            hipDeviceSynchronize();

            // ! Calcular la energía total
            total_ener = 0.0f;
            for (int k = 0; k < n_part; k++)
                total_ener += ener[k];

            if (i % 1000 == 0)
            {
                // for (size_t k = 0; k < n_part; k++)
                // {
                //     printf("%.10f %.10f %.10f\n", x[k], y[k], z[k]);
                //     printf("FORCES\n");
                //     printf("%.10f %.10f %.10f\n", fx[k], fy[k], fz[k]);
                // }
                printf("%d %.10f Thermal\n", i, total_ener / ((float)(n_part)));
            }
            if (i % 100 == 0)
            {
                fprintf(f_ener, "%d %.10f\n", i, total_ener / ((float)(n_part)));
            }
        }
        fclose(f_ener);

        // Guardar la configuración final después de termalizar
        for (int i = 0; i < n_part; i++)
        {
            fprintf(f_final, "%.10f %.10f %.10f %.10f %.10f %.10f\n", x[i], y[i], z[i], fx[i], fy[i], fz[i]);
        }
        fclose(f_final);
    }

    // Calcular la g(r)
    int nprom = 0;
    int ncep = 10;
    for (int i = 0; i < ncp; i++)
    {
        // * Crear números aleatorios
        hiprandGenerateNormal(gen, rngvec_dev, rng_size, 0.0f, 1.0f);
        position<<<bloques, hilos>>>(x, y, z, fx, fy, fz, d_tiempo, l_caja, n_part, 0, rngvec_dev);
        hipDeviceSynchronize();
        rdf_force<<<bloques, hilos>>>(x, y, z, fx, fy, fz, n_part, l_caja, ener);
        hipDeviceSynchronize();

        // ! Calcular la energía total
        total_ener = 0.0f;
        for (int k = 0; k < n_part; k++)
            total_ener += ener[k];

        if (i % 1000 == 0)
        {
            printf("%d %.10f Average\n", i, total_ener / ((float)(n_part)));
        }
        if (i % ncep == 0)
        // if (i%n_part == 0) // Promediar cada numero total de particulas
        {
            t[nprom] = d_tiempo * (float)(ncep) * nprom;
            for (int j = 0; j < n_part; j++)
            {
                cfx[nprom * mp + j] = x[j];
                cfy[nprom * mp + j] = y[j];
                cfz[nprom * mp + j] = z[j];
            }
            nprom++;
            gr(x, y, z, g, n_part, l_caja);
        }
    }

    printf("%.10f %d\n", dr, nprom);

    f_gr = fopen(argv[7], "w");
    float *r;
    hipMallocManaged(&r, nm * sizeof(float));
    float dv = 0.0f;
    float fnorm = 0.0f;

    for (int i = 1; i < nm; i++)
    {
        r[i] = (i - 1) * dr;
        dv = 4.0f * PI * r[i] * r[i] * dr;
        fnorm = powf(l_caja, 3.0f) / (powf(n_part, 2.0f) * nprom * dv);
        g[i] = g[i] * fnorm;
        fprintf(f_gr, "%.10f %.10f\n", r[i], g[i]);
    }
    fclose(f_gr);

    // Mean-square displacement and intermediate scattering function
    difusion(nprom, n_part, cfx, cfy, cfz, wt);

    wt_f = fopen("wt.dat", "w");
    for (int i = 0; i < (ncp / ncep); i++)
    {
        fprintf(wt_f, "%.10f %.10f\n", t[i], wt[i]);
    }
    fclose(wt_f);

    // ! Cleanup
    hiprandDestroyGenerator(gen);
    hipFree(x);
    hipFree(y);
    hipFree(z);
    hipFree(fx);
    hipFree(fy);
    hipFree(fz);
    hipFree(rngvec_dev);
    hipFree(ener);
    hipFree(r);
    hipFree(g);
    hipFree(t);
    hipFree(cfx);
    hipFree(cfy);
    hipFree(cfz);
    hipFree(wt);
    hipDeviceReset();

    return EXIT_SUCCESS;
}
