#include "hip/hip_runtime.h"
#include "functionsl.h"

void iniconf(float *x, float *y, float *z, float rho, float rc, int num_part)
{
    // Definir la distancia según la densidad
    float dist = powf(1.0 / rho, 1.0 / 3.0);

    // Inicializar las primeras posiciones
    x[0] = -rc + (dist / 2.0);
    y[0] = -rc + (dist / 2.0);
    z[0] = -rc + (dist / 2.0);

    for (int i = 1; i < num_part - 1; i++)
    {
        x[i] = x[i - 1] + dist;
        y[i] = y[i - 1];
        z[i] = z[i - 1];

        if (x[i] > rc)
        {
            x[i] = x[0];
            y[i] = y[i - 1] + dist;

            if (y[i] > rc)
            {
                x[i] = x[0];
                y[i] = y[0];
                z[i] = z[i - 1] + dist;
            }
        }
    }
}

// __device__ void hardsphere(float r_pos, float uij, float fij)
// {
//     uij = (a_param / temp) * (powf(1.0f / r_pos, lambda) - powf(1.0f / r_pos, lambda - 1.0f));
//     fij = lambda * powf(1.0f / r_pos, lambda + 1.0f) - (lambda - 1.0f) * powf(1.0f / r_pos, lambda);

//     fij *= (a_param / temp);
//     uij += 1.0f / temp;
// }

__global__
void rdf_force(float *x, float *y, float *z, float *fx, float *fy, float *fz,
                          int num_part, float box_l, float *ener)
{
    // Parámetros
    float rc = box_l * 0.5f;
    // float d_r = rc / nm;

    // Inicializar algunas variables de la posicion
    float xij = 0.0f, yij = 0.0f, zij = 0.0f, rij = 0.0f;
    float fij = 0.0f;
    float uij = 0.0f;
    float potential = 0.0f;
    int i = 0, j = 0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (i = idx; i < num_part; i += stride)
    {
        // Inicializar valores
        potential = 0.0f;
        fx[i] = 0.0f;
        fy[i] = 0.0f;
        fz[i] = 0.0f;
        ener[i] = 0.0f;

        for (j = 0; j < num_part; j++)
        {
            if (i == j)
                continue;
            // Siempre inicializar en cero
            uij = 0.0f;
            fij = 0.0f;

            // Contribucion de pares
            xij = x[j] - x[i];
            yij = y[j] - y[i];
            zij = z[j] - z[i];

            // Condiciones de frontera
            xij -= box_l * roundf(xij / box_l);
            yij -= box_l * roundf(yij / box_l);
            zij -= box_l * roundf(zij / box_l);

            rij = sqrtf(xij * xij + yij * yij + zij * zij);

            if (rij < rc)
            {
                // Siempre se calcula la fuerza
                if (rij < b_param)
                {
                    uij = (a_param / temp) * (powf(1.0f / rij, lambda) - powf(1.0f / rij, lambda - 1.0f));
                    fij = lambda * powf(1.0f / rij, lambda + 1.0f) - (lambda - 1.0f) * powf(1.0f / rij, lambda);
                    fij *= (a_param / temp);
                    uij += (1.0f / temp);
                }
                else
                {
                    uij = 0.0f;
                    fij = 0.0f;
                }

                // Actualizar los valores de las fuerzas
                fx[i] += (fij * xij) / rij;
                fy[i] += (fij * yij) / rij;
                fz[i] += (fij * zij) / rij;

                fx[j] -= (fij * xij) / rij;
                fy[j] -= (fij * yij) / rij;
                fz[j] -= (fij * zij) / rij;

                // Actualizar los valores de la energía
                potential += uij;
                // printf("%f\n", ener);
            }
        }
        ener[i] = potential;
    }
}

void gr(float* x, float* y, float* z, float* g, int num_part, float box_l)
{
    // Parámetros
    float rc = box_l * 0.5f;
    float d_r = rc / nm;

    int nbin = 0;
    int i = 0, j = 0;
    float xij = 0.0f, yij = 0.0f, zij = 0.0f, rij = 0.0f;

    for (i = 0; i < num_part; i++)
    {
        for (j = i+1; j < num_part-1; j++)
        {

            // Contribucion de pares
            xij = x[j] - x[i];
            yij = y[j] - y[i];
            zij = z[j] - z[i];

            // Condiciones de frontera
            xij -= (box_l * round(xij/box_l));
            yij -= (box_l * round(yij/box_l));
            zij -= (box_l * round(zij/box_l));

            rij = sqrtf(xij*xij + yij*yij + zij*zij);

            if (rij < rc)
            {
                nbin = (int)(rij/d_r) + 1;
                if (nbin <= nm)
                {
                    g[nbin] += 2.0f;
                }
            }
        }
    }
}

__global__ void position(float *x, float *y, float *z, float *fx, float *fy, float *fz, float dtt,
                         float box_l, int num_part, int pbc, float *randvec)
{
    // Inicializar algunas variables
    float dx = 0.0f;
    float dy = 0.0f;
    float dz = 0.0f;
    float sigma = sqrtf(2.0f * dtt);
    int i = 0;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    for (i = idx; i < num_part; i += stride)
    {
        dx = sigma * randvec[3 * i];
        dy = sigma * randvec[(3 * i) + 1];
        dz = sigma * randvec[(3 * i) + 2];

        x[i] += fx[i] * dtt + dx;
        y[i] += fy[i] * dtt + dy;
        z[i] += fz[i] * dtt + dz;

        if (pbc == 1)
        {
            x[i] -= box_l * roundf(x[i] / box_l);
            y[i] -= box_l * roundf(y[i] / box_l);
            z[i] -= box_l * roundf(z[i] / box_l);
        }
    }
}

__global__
void difusion(const int nprom, const int n_part, float *cfx, float *cfy, float *cfz, float *wt)
{
    float dif = 0.0f;
    int i = 0, j = 0, k = 0;
    float dx = 0.0f, dy = 0.0f, dz = 0.0f, aux = 0.0f;
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int stride = blockDim.x * gridDim.x;

    // Mean-squared displacement
    for (i = 0; i < nprom; i++)
    {
        dif = 0.0f;
        // printf("%d\n", nprom-i);
        for (j = 0; j < nprom-i; j++)
        {
            // for (k = 0; k < n_part; k++)
            for (k = idx; k < n_part; k += stride)
            {
                dx = cfx[(j+i)*mp + k] - cfx[j*mp + k];
                dy = cfy[(j+i)*mp + k] - cfy[j*mp + k];
                dz = cfz[(j+i)*mp + k] - cfz[j*mp + k];
                dif += dx*dx + dy*dy + dz*dz;
            }
        }
        aux = n_part * (nprom-i);
        wt[i] = dif / aux;
    }
}
